
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void grep(char* buffer, int* len, char* key, int* keylen);

__global__ void search(char* d_in, int* d_out, char* d_key, int* d_len, int* d_keylen){
        //index of the thread is the index of the line it will process
        int i = threadIdx.x;
        //initial result to 0 
        d_out[i] = 0;
        if (i == 0){
                //search each charact in line
                for (int k=0; k < d_len[0] - *d_keylen; k++){
                        if (d_in[k] == d_key[0]) {
                                //compare all charact with keywords
                                for (int j=1; j< *d_keylen; j++) {
                                        if (d_in[k+j] != d_key[j]) {
                                            break;
					}
					//all charact match
                                        if (j == *d_keylen - 1){
                                            //set result to 1 means find match word
                                            d_out[i] = 1;
                                            break;
					}
                                }
                        }
                        //find 1 match is ok to break out
                        if (d_out[i] == 1) {
                                break;
                        }
                }
        }
        else {
                //search each charact in line
                for (int k=d_len[i-1]; k < d_len[i] - *d_keylen; k++) {
                        if (d_in[k] == d_key[0]) {
                                //compare all charact with keywords
                                for (int j=1; j < *d_keylen; j++) {
                                        if (d_in[k+j] != d_key[j]){
                                                break;
					}
					// all charact match
                                        if (j == *d_keylen - 1) {
                                                //set result to 1 means find match word
                                                d_out[i] = 1;
                                                break;
					}
                                }
                        }
                        //find 1 match is ok to break out
                        if (d_out[i] == 1) {
                                break;
                        }
                }
        }
}
unsigned long BUFFER_SIZE = 1000000;
unsigned long BUFFER_BYTES = BUFFER_SIZE * sizeof(char);

void grep(char* buffer, int* len, char* key, int* keylen) {
        // generate the output array on the host
        int h_out[1000];

        // declare GPU memory pointers
        char* d_in;
        int* d_out;
        char* d_key;
        int* d_len;
        int* d_keylen;

        // allocate GPU memory
        hipMalloc((void**) &d_in, BUFFER_BYTES);
        hipMalloc((void**) &d_out, 1000 * sizeof(int));
        hipMalloc((void**) &d_key, *keylen * sizeof(int));
        hipMalloc((void**) &d_len, 1000 * sizeof(int));
        hipMalloc((void**) &d_keylen, sizeof(int));

        // transfer the array to the GPU
        hipMemcpy(d_in, buffer, BUFFER_BYTES, hipMemcpyHostToDevice);
        hipMemcpy(d_key, key, *keylen * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_len, len, 1000 * sizeof(int) , hipMemcpyHostToDevice);
        hipMemcpy(d_keylen, keylen, sizeof(int), hipMemcpyHostToDevice);

        // launch the kernel
        search<<<1, 1000>>>(d_in, d_out, d_key, d_len, d_keylen);

        // copy back the result array to the CPU
        hipMemcpy(h_out, d_out, 1000*sizeof(int), hipMemcpyDeviceToHost);

        // print out the resulting array
        for (int q=0; q < 1000; q++) {
                if (h_out[q] == 1){
                        if (q == 0) {
                                for (int j=0; j < len[0]; j++){
                                        printf("%c", buffer[j]);
                                }
                        }
                        else {
                                for (int j=len[q-1]; j < len[q]; j++){
                                        printf("%c", buffer[j]);
                                }	
                        }
			printf("\n");
                }
        }
        //free GPU memory
        hipFree(d_in);
        hipFree(d_out);
        hipFree(d_key);
        hipFree(d_len);
        hipFree(d_keylen);
}
